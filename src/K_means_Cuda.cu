#include "hip/hip_runtime.h"
//
// Created by marco on 22/05/20.
//


#include "K_means_Cuda.cuh"
#include <cmath>
#include <iostream>



__constant__ short constK;
__constant__ int constNumPoint;
__constant__ short constDimPoint;

void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess) {
        return;
    }
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}



void print_device(double *device, int row, int col){
    double *host;
    host = (double *) malloc(row * col * sizeof(double));
    hipMemcpy(host, device, row * col * sizeof(double),hipMemcpyDeviceToHost);

    for (auto i = 0; i < row; i++) {
        for (auto j = 0; j < col; j++) {
            std::cout <<"- "<< host[i * col + j] << " ";
        }
        std::cout << "-" << std::endl;
    }
    std::cout << std::endl;
}

void print_device(short *device, int row, int col){
    short *host;
    host = (short *) malloc(row * col * sizeof(short));
    hipMemcpy(host, device, row * col * sizeof(short),hipMemcpyDeviceToHost);

    for (auto i = 0; i < row; i++) {
        for (auto j = 0; j < col; j++) {
            std::cout <<"- "<< host[i * col + j] << " ";
        }
        std::cout << "-" << std::endl;
    }
    std::cout << std::endl;
}

void print_device(int *device, int row, int col){
    int *host;
    host = (int *) malloc(row * col * sizeof(int));
    hipMemcpy(host, device, row * col * sizeof(int),hipMemcpyDeviceToHost);

    for (auto i = 0; i < row; i++) {
        for (auto j = 0; j < col; j++) {
            std::cout <<"- "<< host[i * col + j] << " ";
        }
        std::cout << "-" << std::endl;
    }
    std::cout << std::endl;
}



/*
//INIZIALIZE CENDROID ASSIGNEMENT TO ZERO FOR ALL POINT'S DATASETS
//Assegno ogni punto al cluster -1
__global__
void initialize_assignment(short * deviceAssignment){
    unsigned int threadId = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (threadId < constNumPoint){
        //printf("STAMPA DEL DEVICEASSIGNEMENT [%d] \n",deviceAssignment[threadId]);
        deviceAssignment[threadId] = -1;

    }
}
*/


__device__ double doubleAtomicAdd(double*address, double val){
    auto *address_as_ull = (unsigned long long int *) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val + __longlong_as_double((long long int)assumed)));
    } while (assumed != old);
    return __longlong_as_double((long long int)old);
}


__host__
bool checkEqualAssignment(const short * hostOldAssignment, const short * hostAssignment, const int numPoint){
    for (auto i = 0; i < numPoint; i++){
        if(hostOldAssignment[i] != hostAssignment[i]){
            return false;
        }
    }
    return true;
}


__global__
void compute_distances(const double * deviceDataset, const double * deviceCentroids, double * deviceDistances){
    double distance = 0;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < constNumPoint && col < constK){
        for (int i = 0; i < constDimPoint; i++) {
            distance += pow(deviceDataset[row*constDimPoint+i] - deviceCentroids[col*constDimPoint+i], 2);
            //printf("centroide %f ",deviceCentroids[col*constDimPoint+i]);
        }
        deviceDistances[row*constK+col] = sqrt(distance);
        //if (deviceDistances[row*constK+col] == 0){
        //    printf("distanza %f ",deviceDistances[row*constK+col]);
       //}
    }
}


__global__
void point_assignment(const double *deviceDistances, short *deviceAssignment){
    unsigned int threadId = (blockDim.x * blockIdx.x) + threadIdx.x;
    double min = INFINITY;
    short clusterLabel;
    double distance;
    if (threadId < constNumPoint){
        for (auto i = 0; i < constK; i++){
            distance = deviceDistances[threadId*constK + i];
            //printf("distanza %f ",distance);
            if(distance < min){
                min = distance;
                clusterLabel = i;
            }
        }
        deviceAssignment[threadId] = clusterLabel;
        //printf(" clusterID: %d",deviceAssignment[threadId]);
    }
}

__global__
void initialize_centroids(double * deviceCentroids){
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < constDimPoint && row < constK){
        deviceCentroids[row*constDimPoint + col] = 0;
    }
}


//Original compute sum with 2D grid (better with dataset with too much dimensions)
__global__
void compute_sum(const double *deviceDataset, double * deviceCentroids, const short *deviceAssignment, int * deviceCount){
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < constDimPoint && row < constNumPoint){
        short clusterId = deviceAssignment[row];
        //printf(" clusterID: %d",clusterId);
        doubleAtomicAdd(&deviceCentroids[clusterId*constDimPoint +col], deviceDataset[row*constDimPoint +col]);
        atomicAdd(&deviceCount[clusterId], 1);
        //printf(" c %f ",clusterId);
    }
}

//compute sum with 1D grid and iterate on point's dimensions
__global__
void compute_sum2(const double *deviceDataset, double * deviceCentroids, const short *deviceAssignment, int * deviceCount){
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < constNumPoint){
        short clusterId = deviceAssignment[row];
        for (auto i = 0; i< constDimPoint; i++){
            doubleAtomicAdd(&deviceCentroids[clusterId*constDimPoint+i], deviceDataset[row*constDimPoint+i]);
        }
        atomicAdd(&deviceCount[clusterId], 1);
    }
}

//Update centroids with 2D grid (better with dataset with too much dimensions)
__global__
void update_centroids(double * deviceCentroids, const int * deviceCount){
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < constDimPoint && row < constK) {
        //printf(" count  %f", (deviceCount[row]));
        deviceCentroids[row * constDimPoint + col] = deviceCentroids[row * constDimPoint + col] / (double(deviceCount[row])/constDimPoint);
        //printf(" centroide %f ",deviceCount[row]);
    }
}

//Update centroids with 1D grid (no need to divide count for point's dimensions)
__global__
void update_centroids2(double * deviceCentroids, const int * deviceCount){
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < constDimPoint && row < constK) {
        deviceCentroids[row * constDimPoint + col] /= deviceCount[row];

    }
}




__host__
std::tuple<double *, short *>cuda_KMeans(double * deviceDataset, double * deviceCentroids, const int numPoint, const short k, const short dimPoint){
    //int c = 0;
    dim3 dimBlockDistance(2, 512, 1);
    dim3 dimGridDistance(ceil(k/2.0), ceil(numPoint/512.0), 1);

    dim3 dimBlockInitialize(16, 16, 1);
    dim3 dimGridInitialize(ceil(dimPoint / 16.0), ceil(k / 16.0), 1);

    dim3 dimBlockComputeSum(2, 512, 1);
    dim3 dimGridComputeSum(ceil(dimPoint / 2.0), ceil(numPoint / 512.0), 1);

    dim3 dimBlockUpdateCentroids(16, 16, 1);
    dim3 dimGridUpdateCentroids(ceil(dimPoint / 16.0), ceil(k / 16.0), 1);

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(constK), &k, sizeof(short)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(constNumPoint), &numPoint, sizeof(int)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(constDimPoint), &dimPoint, sizeof(short)));
    //constant_dimPoint = dimPoint;
    bool convergence = false;

    short * hostOldAssignment;
    hostOldAssignment = (short *) malloc(numPoint * sizeof(short));

    short * hostAssignment;
    hostAssignment = (short *) malloc(numPoint * sizeof(short));


    short * deviceAssignment;
    CUDA_CHECK_RETURN(hipMalloc((void **) &deviceAssignment, numPoint*sizeof(short)));
    double * deviceDistances;
    CUDA_CHECK_RETURN(hipMalloc((void**) &deviceDistances, numPoint*k*sizeof(double)));
    int * deviceCount;
    CUDA_CHECK_RETURN(hipMalloc((void**) &deviceCount, k*sizeof(int)));



    while (!convergence){
        //ASSIGNMENT
        //Find the nearest centroid and assign the point to that cluster
        compute_distances<<<dimGridDistance, dimBlockDistance>>>(deviceDataset, deviceCentroids, deviceDistances);
        hipDeviceSynchronize();
        point_assignment<<<ceil(numPoint/1024.0), 1024>>>(deviceDistances, deviceAssignment);
        hipDeviceSynchronize();

        //CENTROIDS UPDATE
        //Initialize centroids to 0 and set count to 0 (for compute means)
        initialize_centroids<<<dimGridInitialize, dimBlockInitialize>>>(deviceCentroids);

        //print_device(deviceCentroids, k,  dimPoint);
        //return{deviceCentroids, hostAssignment};
        CUDA_CHECK_RETURN(hipMemset(deviceCount, 0, k*sizeof(int)));
        //print_device(deviceCount, k,  1);
        hipDeviceSynchronize();
        //Compute all sum for centroids

        compute_sum<<<dimGridComputeSum,dimBlockComputeSum>>>(deviceDataset, deviceCentroids, deviceAssignment, deviceCount);
        //compute_sum2<<<ceil(numPoint/1024.0), 1024>>>(deviceDataset, deviceCentroids, deviceAssignment, deviceCount);

        hipDeviceSynchronize();
        //printf("\n STAMPA DI TEST \n");
        //print_device(deviceCentroids, k,  dimPoint);
        //printf("\n");
        //return{deviceCentroids, hostAssignment};
        //Compute mean: division for count


        update_centroids<<<dimGridUpdateCentroids,dimBlockUpdateCentroids>>>(deviceCentroids,deviceCount);
        //update_centroids2<<<dimGridUpdateCentroids,dimBlockUpdateCentroids>>>(deviceCentroids,deviceCount);

        hipDeviceSynchronize();

        CUDA_CHECK_RETURN(hipMemcpy(hostAssignment, deviceAssignment, numPoint*sizeof(short), hipMemcpyDeviceToHost));


/*
        std::cout << "PRINT HOST ASSIGNMENT     ";
        for(auto i = 0; i<numPoint; i++){
            std::cout << hostAssignment[i] << " ";
        }
        std::cout << "\n" ;

        std::cout << "PRINT HOST OLD ASSIGNMENT ";
        for(auto i = 0; i<numPoint; i++){
            std::cout << hostOldAssignment[i] << " " ;
        }
        std::cout << "\n" ;
*/
        //c ++;


        if (checkEqualAssignment(hostOldAssignment, hostAssignment, numPoint)){
            convergence = true;
        }
        else{
            CUDA_CHECK_RETURN(hipMemcpy(hostOldAssignment, deviceAssignment, numPoint*sizeof(short), hipMemcpyDeviceToHost));
        }
        //printf("\n");

    }
    //std::cout << "Numero di iterazioni: " << c << " \n";

    return{deviceCentroids, hostAssignment};
}
